#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// PTX code as a string
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry vector_add_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r13, %r4, %r18, %r21, %r5, %r9, %r6, %r7, %r22, %r8, %r12;
    .reg .u64 %rd1, %rd16, %rd2, %rd17, %rd20, %rd15, %rd14, %rd19, %rd3, %rd0;
    .reg .pred %p10, %p11;
    ld.param.u64 %rd0, [param9];
    ld.param.u64 %rd1, [param10];
    ld.param.u64 %rd2, [param11];
    ld.param.u64 %rd3, [param12];
    block_1_start:
    mov.u32 %r4, %ntid.x;
    mov.u32 %r5, %ctaid.x;
    mul.lo.s32 %r6, %r4, %r5;
    mov.u32 %r7, %tid.x;
    add.s32 %r8, %r6, %r7;
    mov.u32 %r5, %r8;
    cvt.u32.u64 %r9, %rd3;
    setp.ge.u32 %p10, %r8, %r9;
    not.pred %p11, %p10;
    @%p11 bra br_if_label_0;
    br_if_label_0:
    mov.u32 %r12, 2;
    shl.b32 %r13, %r5, %r12;
    mov.u32 %r5, %r13;
    cvt.u64.u32 %rd14, %r13;
    add.s64 %rd15, %rd2, %rd14;
    cvt.u64.u32 %rd16, %r5;
    add.s64 %rd17, %rd1, %rd16;
    ld.global.u32 %r18, [%rd17];
    cvt.u64.u32 %rd19, %r5;
    add.s64 %rd20, %rd0, %rd19;
    ld.global.u32 %r21, [%rd20];
    add.s32 %r22, %r18, %r21;
    st.global.u32 [%rd15], %r22;
}

)ptx";

hipError_t checkCudaError(hipError_t result, const char* message) {
    if (result != hipSuccess) {
        const char* errorString;
        hipDrvGetErrorString(result, &errorString);
        std::cerr << message << ": " << errorString << std::endl;
    }
    return result;
}


int main() {
    // Initialize CUDA
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);

    // Load the PTX code as a module
    hipError_t res = hipModuleLoadData(&cuModule, ptx_code);
    if (res != hipSuccess) {
        std::cerr << "Error loading PTX code." << std::endl;
        return -1;
    }

    // Get the kernel function
    res = hipModuleGetFunction(&cuFunction, cuModule, "vector_add_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting kernel function." << std::endl;
        return -1;
    }

    // Vector size
    int N = 1024;
    const int size = N * sizeof(int);

    // Allocate host memory
    std::vector<int> h_A(N, 1); // Vector A initialized to 1
    std::vector<int> h_B(N, 2); // Vector B initialized to 2
    std::vector<int> h_C(N, 0); // Result vector C

    // Allocate device memory
    hipDeviceptr_t d_A, d_B, d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpyHtoD(d_A, h_A.data(), size);
    hipMemcpyHtoD(d_B, h_B.data(), size);
    checkCudaError(hipMemcpyHtoD(d_A, h_A.data(), size), "Error copying h_A to device");
    checkCudaError(hipMemcpyHtoD(d_B, h_B.data(), size), "Error copying h_B to device");


    std::vector<int> h_A_check(N, 0);
    std::vector<int> h_B_check(N, 0);
    std::vector<int> h_C_check(N, 0);

    hipMemcpyDtoH(h_A_check.data(), d_A, size);
    hipMemcpyDtoH(h_B_check.data(), d_B, size);
    hipMemcpyDtoH(h_C_check.data(), d_C, size);

    std::cout << "Device vector A (copied back):" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_A_check[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    std::cout << "Device vector B (copied back):" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_B_check[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    std::cout << "Device vector C (copied back):" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_C_check[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Prepare kernel parameters: must match param0..param3 in PTX
    void* args[] = {
        &d_A,       // param0
        &d_B,       // param1
        &d_C,       // param2
        &N          // param3
    };


    // Print input vector A
    std::cout << "Input vector A:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_A[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Print input vector B
    std::cout << "Input vector B:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_B[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t launchResult = hipModuleLaunchKernel(
        cuFunction,
        blocksPerGrid, 1, 1,       // Grid dimensions
        threadsPerBlock, 1, 1,     // Block dimensions
        0, nullptr,                // Shared memory and stream
        args, nullptr              // Kernel arguments
    );


    std::cout << "Kernel parameters:" << std::endl;
    std::cout << "d_A: " << d_A << std::endl;
    std::cout << "d_B: " << d_B << std::endl;
    std::cout << "d_C: " << d_C << std::endl;
    std::cout << "N: " << N << std::endl;

    if (launchResult != hipSuccess) {
        std::cerr << "Kernel launch failed with error code: " << launchResult << std::endl;
    } else {
        std::cout << "Kernel launched successfully." << std::endl;
    }

    // Copy the result back to the host
    hipMemcpyDtoH(h_C.data(), d_C, size);

    // Print the result
    std::cout << "Result vector C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % 16 == 0) { // Print 16 values per line for readability
            std::cout << std::endl;
        }
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}