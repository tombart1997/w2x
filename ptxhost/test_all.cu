#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>


const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry vector_add_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r2, %r3, %r8, %r0, %r17, %r21, %r1, %r20, %r4, %r7;
    .reg .u64 %rd14, %rd15, %rd11, %rd12, %rd16, %rd19, %rd13, %rd9, %rd18, %rd10, %rd5;
    .reg .pred %p6;
    vector_add_kernel_start:
    ld.param.u64 %rd9, [param9];
    ld.param.u64 %rd10, [param10];
    ld.param.u64 %rd11, [param11];
    ld.param.u64 %rd12, [param12];
    block_1_start:
    mov.u32 %r0, %ntid.x;
    mov.u32 %r1, %ctaid.x;
    mul.lo.s32 %r2, %r0, %r1;
    mov.u32 %r3, %tid.x;
    add.u32 %r4, %r2, %r3;
    mov.u32 %r1, %r4;
    cvt.u64.u32 %rd5, %r4;
    setp.ge.u64 %p6, %rd5, %rd12;
    @%p6 bra block_1_end;
    mov.u32 %r7, 2;
    shl.b32 %r8, %r1, %r7;
    mov.u32 %r1, %r8;
    cvt.u64.u32 %rd13, %r8;
    add.u64 %rd14, %rd11, %rd13;
    cvt.u64.u32 %rd15, %r1;
    add.u64 %rd16, %rd10, %rd15;
    ld.global.u32 %r17, [%rd16];
    cvt.u64.u32 %rd18, %r1;
    add.u64 %rd19, %rd9, %rd18;
    ld.global.u32 %r20, [%rd19];
    add.u32 %r21, %r17, %r20;
    st.global.u32 [%rd14], %r21;
    block_1_end:
    vector_add_kernel_end:
}

.visible .entry vector_add_loop_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r3, %r1, %r6, %r28, %r7, %r8, %r4, %r18, %r24, %r15, %r13, %r16, %r14, %r2, %r0, %r17, %r27, %r5, %r29, %r30;
    .reg .u64 %rd22, %rd20, %rd9, %rd10, %rd11, %rd25, %rd12, %rd21, %rd23, %rd26;
    .reg .pred %p19;
    vector_add_loop_kernel_start:
    ld.param.u64 %rd9, [param9];
    ld.param.u64 %rd10, [param10];
    ld.param.u64 %rd11, [param11];
    ld.param.u64 %rd12, [param12];
    mov.u32 %r0, %ntid.x;
    mov.u32 %r1, %ntid.z;
    mul.lo.s32 %r2, %r0, %r1;
    mov.u32 %r3, %ntid.y;
    mul.lo.s32 %r4, %r2, %r3;
    mov.u32 %r3, %r4;
    mov.u32 %r5, 2;
    shl.b32 %r6, %r4, %r5;
    mov.u32 %r7, %r6;
    mov.u32 %r8, %ctaid.x;
    mul.lo.s32 %r13, %r0, %r8;
    mov.u32 %r14, %tid.x;
    add.u32 %r15, %r13, %r14;
    mov.u32 %r1, %r15;
    mov.u32 %r16, 2;
    shl.b32 %r17, %r15, %r16;
    mov.u32 %r0, %r17;
    loop_0_start:
    block_2_start:
    cvt.u32.u64 %r18, %rd12;
    setp.lt.u32 %p19, %r1, %r18;
    @%p19 bra block_2_end;
    ret;
    block_2_end:
    cvt.u64.u32 %rd20, %r0;
    add.u64 %rd21, %rd11, %rd20;
    cvt.u64.u32 %rd22, %r0;
    add.u64 %rd23, %rd10, %rd22;
    ld.global.u32 %r24, [%rd23];
    cvt.u64.u32 %rd25, %r0;
    add.u64 %rd26, %rd9, %rd25;
    ld.global.u32 %r27, [%rd26];
    add.u32 %r28, %r24, %r27;
    st.global.u32 [%rd21], %r28;
    add.u32 %r29, %r0, %r7;
    mov.u32 %r0, %r29;
    add.u32 %r30, %r1, %r3;
    mov.u32 %r1, %r30;
    bra loop_0_start;
    loop_0_end:
    vector_add_loop_kernel_end:
}

.visible .entry matrix_mul_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12,
    .param .u64 param13,
    .param .u64 param14
) {
    .reg .u32 %r1, %r45, %r38, %r34, %r49, %r52, %r24, %r3, %r51, %r16, %r30, %r53, %r42, %r23, %r8, %r2, %r50, %r4, %r18, %r41, %r21, %r17, %r15, %r0, %r5;
    .reg .u64 %rd46, %rd55, %rd6, %rd29, %rd9, %rd22, %rd36, %rd12, %rd39, %rd32, %rd13, %rd19, %rd25, %rd10, %rd40, %rd54, %rd33, %rd28, %rd37, %rd48, %rd44, %rd56, %rd27, %rd43, %rd47, %rd14, %rd31, %rd11, %rd26;
    .reg .pred %p20, %p7, %p35;
    matrix_mul_kernel_start:
    ld.param.u64 %rd9, [param9];
    ld.param.u64 %rd10, [param10];
    ld.param.u64 %rd11, [param11];
    ld.param.u64 %rd12, [param12];
    ld.param.u64 %rd13, [param13];
    ld.param.u64 %rd14, [param14];
    block_3_start:
    mov.u32 %r0, %ntid.y;
    mov.u32 %r1, %ctaid.y;
    mul.lo.s32 %r2, %r0, %r1;
    mov.u32 %r3, %tid.y;
    add.u32 %r4, %r2, %r3;
    mov.u32 %r5, %r4;
    cvt.u64.u32 %rd6, %r4;
    setp.ge.u64 %p7, %rd6, %rd12;
    @%p7 bra block_3_end;
    mov.u32 %r8, %ntid.x;
    mov.u32 %r15, %ctaid.x;
    mul.lo.s32 %r16, %r8, %r15;
    mov.u32 %r17, %tid.x;
    add.u32 %r18, %r16, %r17;
    mov.u32 %r15, %r18;
    cvt.u64.u32 %rd19, %r18;
    setp.ge.u64 %p20, %rd19, %rd14;
    @%p20 bra block_3_end;
    mov.u32 %r21, 2;
    shl.b64 %rd22, %rd14, %r21;
    mov.u64 %rd12, %rd22;
    mov.u32 %r23, 2;
    shl.b32 %r24, %r15, %r23;
    cvt.u64.u32 %rd25, %r24;
    add.u64 %rd26, %rd10, %rd25;
    mov.u64 %rd27, %rd26;
    cvt.u64.u32 %rd28, %r5;
    mul.lo.s64 %rd29, %rd28, %rd13;
    mov.u32 %r30, 2;
    shl.b64 %rd31, %rd29, %r30;
    add.u64 %rd32, %rd9, %rd31;
    mov.u64 %rd33, %rd32;
    mov.u32 %r34, 0;
    mov.u32 %r3, %r34;
    loop_0_start:
    block_4_start:
    setp.ne.u64 %p35, %rd13, 0;
    @%p35 bra block_4_end;
    cvt.u64.u32 %rd36, %r5;
    mul.lo.s64 %rd37, %rd36, %rd14;
    mov.u32 %r38, 2;
    shl.b64 %rd39, %rd37, %r38;
    add.u64 %rd40, %rd11, %rd39;
    mov.u32 %r41, 2;
    shl.b32 %r42, %r15, %r41;
    cvt.u64.u32 %rd43, %r42;
    add.u64 %rd44, %rd40, %rd43;
    st.global.u32 [%rd44], %r3;
    bra block_3_end;
    block_4_end:
    mov.s32 %r45, -1;
    cvt.s64.s32 %rd46, %r45;
    add.s64 %rd47, %rd13, %rd46;
    cvt.u64.s64 %rd48, %rd47;
    mov.u64 %rd13, %rd48;
    ld.global.u32 %r49, [%rd27];
    ld.global.u32 %r50, [%rd33];
    mul.lo.s32 %r51, %r49, %r50;
    add.u32 %r52, %r51, %r3;
    mov.u32 %r3, %r52;
    mov.u32 %r53, 4;
    cvt.u64.u32 %rd54, %r53;
    add.u64 %rd55, %rd33, %rd54;
    mov.u64 %rd33, %rd55;
    add.u64 %rd56, %rd27, %rd12;
    mov.u64 %rd27, %rd56;
    bra loop_0_start;
    loop_0_end:
    block_3_end:
    matrix_mul_kernel_end:
}
)ptx";




int main() {

    //Initialize CUDA Driver API
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);



    // Initalize Matrix for Matrix Mul Kernel
	int h_A[4][4] = {
        {1, 2, 3, 4},
        {5, 6, 7, 8},
        {9, 10, 11, 12},
        {13, 14, 15, 16}
        };
    
    int h_B[4][4] = {
        {16, 15, 14, 13},
        {12, 11, 10, 9},
        {8, 7, 6, 5},
        {4, 3, 2, 1}
    };

    int h_C[4][4] = {0};

    // Flatten h_A and h_B for device transfer
	int flat_A[16];
	int flat_B[16];
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j < 4; ++j) {
			flat_A[i * 4 + j] = h_A[i][j];
			flat_B[i * 4 + j] = h_B[i][j];
		}
	}

    // Initialize Vectors for Vector Add Kernels
    int N = 1024;
    const int size = N * sizeof(int);
    std::vector<int> h_v_A(N, 1); // Vector A initialized to 1
    std::vector<int> h_v_B(N, 2); // Vector B initialized to 2
    std::vector<int> h_v_C(N, 0); // Result vector C
    std::vector<int> h_v_C_l(N, 0); // Result vector C


    // Allocate device memory
    // for Vectors
    hipDeviceptr_t d_A, d_B, d_C, d_A_l, d_B_l, d_C_l, d_D, d_E, d_F;


    
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMalloc(&d_A_l, size);
    hipMalloc(&d_B_l, size);
    hipMalloc(&d_C_l, size);
    // for Matrices
	hipMalloc(&d_D, 16 * sizeof(int));
	hipMalloc(&d_E, 16 * sizeof(int));
	hipMalloc(&d_F, 16 * sizeof(int));


    // Copy data to device
    hipMemcpyHtoD(d_A, h_v_A.data(), size);
    hipMemcpyHtoD(d_B, h_v_B.data(), size);
    hipMemcpyHtoD(d_A_l, h_v_A.data(), size);
    hipMemcpyHtoD(d_B_l, h_v_B.data(), size);
	hipMemcpyHtoD(d_D, flat_A, 16 * sizeof(int));
	hipMemcpyHtoD(d_E, flat_B, 16 * sizeof(int));

    // Load PTX and get kernels one by one
    hipError_t res = hipModuleLoadData(&cuModule, ptx_code);
    hipFunction_t v_add_cu, v_add_loop_cu, m_mul_cu;

    if (res != hipSuccess) {
        std::cerr << "Error loading PTX code: " << res << std::endl;
        const char* errStr;
        hipDrvGetErrorString(res, &errStr);
        std::cerr << "CUDA Error: " << (errStr ? errStr : "Unknown") << std::endl;
        return -1;
    }
    res = hipModuleGetFunction(&v_add_cu, cuModule, "vector_add_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting vector_add_kernel function." << std::endl;
        return -1;
    }
    res = hipModuleGetFunction(&v_add_loop_cu, cuModule, "vector_add_loop_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting vector_add_loop_kernel function." << std::endl;
        return -1;
    }
    res = hipModuleGetFunction(&m_mul_cu, cuModule, "matrix_mul_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting matrix_mul_kernel function." << std::endl;
        return -1;
    }

    // Prepare kernel parameters
    int m = 4, n = 4, p = 4;
    uint64_t m64 = m, n64 = n, p64 = p;
    void* args_v[] = { &d_A, &d_B, &d_C, &N};
    void* args_v_l[] = { &d_A_l, &d_B_l, &d_C_l, &N};
    void* args_m[] = { &d_D, &d_E, &d_F, &m64, &n64, &p64 };
    // Launch configuration for for vectors
	int threadsPerBlock = 1;
	int blocksPerGrid =  1;
    //  matrix multiplication
    int threadsPerBlockX = 16, threadsPerBlockY = 16;
    int blocksPerGridX = (p + threadsPerBlockX - 1) / threadsPerBlockX;
    int blocksPerGridY = (m + threadsPerBlockY - 1) / threadsPerBlockY;


    hipError_t v_add_launch_res = hipModuleLaunchKernel(
        v_add_cu,
        blocksPerGrid, 1, 1,       
        threadsPerBlock, 1, 1,    
        0, nullptr,                
        args_v, nullptr              
    );

    hipCtxSynchronize();

    hipError_t v_add_loop_launch_res = hipModuleLaunchKernel(
        v_add_loop_cu,
        blocksPerGrid, 1, 1,       
        threadsPerBlock, 1, 1,    
        0, nullptr,                
        args_v_l, nullptr              
    );

    hipCtxSynchronize();

    hipError_t m__mul_launch_res = hipModuleLaunchKernel(
        m_mul_cu,
        blocksPerGridX, blocksPerGridY, 1,
        threadsPerBlockX, threadsPerBlockY, 1,
        0, nullptr,
        args_m, nullptr
    );

    hipCtxSynchronize();


    if (v_add_launch_res != hipSuccess) {
        std::cerr << "Vector Add launch failed with error code: " << v_add_launch_res << std::endl;
    } else {
        std::cout << "Vector Add launched successfully." << std::endl;
    }

    if (v_add_loop_launch_res != hipSuccess) {
        std::cerr << "Vector Loop Add launch failed with error code: " << v_add_loop_launch_res << std::endl;
    } else {
        std::cout << "Vector Loop Add launched successfully." << std::endl;
    }

    if (m__mul_launch_res != hipSuccess) {
        std::cerr << "Matrix Multiplication launch failed with error code: " << m__mul_launch_res << std::endl;
    } else {
        std::cout << "Matrix Multiplication launched successfully." << std::endl;
    }

    // Copy the result back to the host
    int flat_C[16];
	hipCtxSynchronize();
    hipMemcpyDtoH(h_v_C.data(), d_C, size);
    hipMemcpyDtoH(h_v_C_l.data(), d_C_l, size);
    hipMemcpyDtoH(flat_C, d_F, 16 * sizeof(int));

    //Print the result of Vector Add Kernel
    std::cout << "Result of Vector Add Kernel:" << std::endl;   
    for (int i = 0; i < N; ++i) {
        std::cout << h_v_C[i] << " ";
    }
    std::cout << std::endl;

    //Print the result of Vector Add Loop Kernel
    std::cout << "Result of Vector Add Loop Kernel:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_v_C_l[i] << " ";
    }
    std::cout << std::endl;

    // Reconstruct h_C from flat_C
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j < 4; ++j) {
			h_C[i][j] = flat_C[i * 4 + j];
		}
	}
    //Print the result of Matrix Multiplication Kernel
    for (int i = 0; i < 4; ++i) {
        for (int j = 0; j < 4; ++j) {
            std::cout << h_C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    //Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_l);
    hipFree(d_B_l);
    hipFree(d_C_l);
    hipFree(d_D);
    hipFree(d_E);
    hipFree(d_F);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);
    return 0;
}