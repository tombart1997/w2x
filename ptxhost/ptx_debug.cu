#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// PTX code as a string
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry vector_add_loop_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r0, %r1, %r2, %r8, %r30, %r7, %r17, %r24, %r28, %r13, %r5, %r14, %r15, %r3, %r16, %r29, %r27, %r4, %r6;
    .reg .u64 %rd25, %rd9, %rd12, %rd11, %rd23, %rd10, %rd21, %rd18, %rd26, %rd22, %rd20;
    .reg .pred %p19;
    vector_add_loop_kernel_start:
    ld.param.u64 %rd9, [param9];
    ld.param.u64 %rd10, [param10];
    ld.param.u64 %rd11, [param11];
    ld.param.u64 %rd12, [param12];
    mov.u32 %r0, %ntid.x;
    mov.u32 %r1, %ntid.z;
    mul.lo.s32 %r2, %r0, %r1;
    mov.u32 %r3, %ntid.y;
    mul.lo.s32 %r4, %r2, %r3;
    mov.u32 %r3, %r4;
    mov.u32 %r5, 2;
    shl.b32 %r6, %r4, %r5;
    mov.u32 %r7, %r6;
    mov.u32 %r8, %ctaid.x;
    mul.lo.s32 %r13, %r0, %r8;
    mov.u32 %r14, %tid.x;
    add.u32 %r15, %r13, %r14;
    mov.u32 %r1, %r15;
    mov.u32 %r16, 2;
    shl.b32 %r17, %r15, %r16;
    mov.u32 %r0, %r17;
    loop_0_start:
    block_2_start:
    cvt.u64.u32 %rd18, %r1;
    setp.lt.u64 %p19, %rd18, %rd12;
    @%p19 bra block_2_end;
    ret;
    block_2_end:
    cvt.u64.u32 %rd20, %r0;
    add.u64 %rd21, %rd11, %rd20;
    cvt.u64.u32 %rd22, %r0;
    add.u64 %rd23, %rd10, %rd22;
    ld.global.u32 %r24, [%rd23];
    cvt.u64.u32 %rd25, %r0;
    add.u64 %rd26, %rd9, %rd25;
    ld.global.u32 %r27, [%rd26];
    add.u32 %r28, %r24, %r27;
    st.global.u32 [%rd21], %r28;
    add.u32 %r29, %r0, %r7;
    mov.u32 %r0, %r29;
    add.u32 %r30, %r1, %r3;
    mov.u32 %r1, %r30;
    bra loop_0_start;
    loop_0_end:
    vector_add_loop_kernel_end:
}
)ptx";

hipError_t checkCudaError(hipError_t result, const char* message) {
    if (result != hipSuccess) {
        const char* errorString;
        hipDrvGetErrorString(result, &errorString);
        std::cerr << message << ": " << errorString << std::endl;
    }
    return result;
}


int main() {
    // Initialize CUDA
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);


    const unsigned int logSize = 8192;
    char error_log[logSize];
    char info_log[logSize];

    hipJitOption options[] = { hipJitOptionErrorLogBuffer, hipJitOptionErrorLogBufferSizeBytes,
                           hipJitOptionInfoLogBuffer, hipJitOptionInfoLogBufferSizeBytes,
                           hipJitOptionLogVerbose };
    void* optionValues[] = { (void*)error_log, (void*)(long)logSize,
                         (void*)info_log, (void*)(long)logSize,
                         (void*)1 };

    hipError_t res = hipModuleLoadDataEx(&cuModule, ptx_code,
                                  sizeof(options)/sizeof(hipJitOption),
                                  options, optionValues);

    if (res != hipSuccess) {
        const char* errStr;
        hipDrvGetErrorString(res, &errStr);
        std::cerr << "Error loading PTX code: " << errStr << "\n";
        std::cerr << "PTX JIT Compilation error log:\n" << error_log << "\n";
        std::cerr << "PTX JIT Compilation info log:\n" << info_log << "\n";
        return -1;
    }


    // Get the kernel function
    res = hipModuleGetFunction(&cuFunction, cuModule, "vector_add_loop_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting kernel function." << std::endl;
        return -1;
    }

    // Vector size
    int N = 1024;
    const int size = N * sizeof(int);

    // Allocate host memory
    std::vector<int> h_A(N, 1); // Vector A initialized to 1
    std::vector<int> h_B(N, 2); // Vector B initialized to 2
    std::vector<int> h_C(N, 0); // Result vector C

    // Allocate device memory
    hipDeviceptr_t d_A, d_B, d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpyHtoD(d_A, h_A.data(), size);
    hipMemcpyHtoD(d_B, h_B.data(), size);
    checkCudaError(hipMemcpyHtoD(d_A, h_A.data(), size), "Error copying h_A to device");
    checkCudaError(hipMemcpyHtoD(d_B, h_B.data(), size), "Error copying h_B to device");

    // Prepare kernel parameters: must match param0..param3 in PTX
    void* args[] = {
        &d_A,       // param0
        &d_B,       // param1
        &d_C,       // param2
        &N,          // param3
    };


    // Print input vector A
    std::cout << "Input vector A:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_A[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Print input vector B
    std::cout << "Input vector B:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_B[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Launch the kernel
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;

    hipError_t launchResult = hipModuleLaunchKernel(
        cuFunction,
        blocksPerGrid, 1, 1,       // Grid dimensions (X, Y, Z)
        threadsPerBlock, 1, 1,     // Block dimensions (X, Y, Z)
        0, nullptr,                // Shared memory and stream
        args, nullptr              // Kernel arguments
    );

    std::cout << "Kernel parameters:" << std::endl;
    std::cout << "d_A: " << d_A << std::endl;
    std::cout << "d_B: " << d_B << std::endl;
    std::cout << "d_C: " << d_C << std::endl;
    std::cout << "N: " << N << std::endl;

    if (launchResult != hipSuccess) {
        std::cerr << "Kernel launch failed with error code: " << launchResult << std::endl;
    } else {
        std::cout << "Kernel launched successfully." << std::endl;
    }

    // Copy the result back to the host
    hipMemcpyDtoH(h_C.data(), d_C, size);

    // Print the result
    std::cout << "Result vector C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % 16 == 0) { // Print 16 values per line for readability
            std::cout << std::endl;
        }
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}