#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// PTX code as a string
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry vector_add_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r2, %r3, %r8, %r0, %r17, %r21, %r1, %r20, %r4, %r7;
    .reg .u64 %rd14, %rd15, %rd11, %rd12, %rd16, %rd19, %rd13, %rd9, %rd18, %rd10, %rd5;
    .reg .pred %p6;
    vector_add_kernel_start:
    ld.param.u64 %rd9, [param9];
    ld.param.u64 %rd10, [param10];
    ld.param.u64 %rd11, [param11];
    ld.param.u64 %rd12, [param12];
    block_1_start:
    mov.u32 %r0, %ntid.x;
    mov.u32 %r1, %ctaid.x;
    mul.lo.s32 %r2, %r0, %r1;
    mov.u32 %r3, %tid.x;
    add.u32 %r4, %r2, %r3;
    mov.u32 %r1, %r4;
    cvt.u64.u32 %rd5, %r4;
    setp.ge.u64 %p6, %rd5, %rd12;
    @%p6 bra block_1_end;
    mov.u32 %r7, 2;
    shl.b32 %r8, %r1, %r7;
    mov.u32 %r1, %r8;
    cvt.u64.u32 %rd13, %r8;
    add.u64 %rd14, %rd11, %rd13;
    cvt.u64.u32 %rd15, %r1;
    add.u64 %rd16, %rd10, %rd15;
    ld.global.u32 %r17, [%rd16];
    cvt.u64.u32 %rd18, %r1;
    add.u64 %rd19, %rd9, %rd18;
    ld.global.u32 %r20, [%rd19];
    add.u32 %r21, %r17, %r20;
    st.global.u32 [%rd14], %r21;
    block_1_end:
    vector_add_kernel_end:
}
)ptx";

hipError_t checkCudaError(hipError_t result, const char* message) {
    if (result != hipSuccess) {
        const char* errorString;
        hipDrvGetErrorString(result, &errorString);
        std::cerr << message << ": " << errorString << std::endl;
    }
    return result;
}


int main() {
    // Initialize CUDA
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);


    const unsigned int logSize = 8192;
    char error_log[logSize];
    char info_log[logSize];

    hipJitOption options[] = { hipJitOptionErrorLogBuffer, hipJitOptionErrorLogBufferSizeBytes,
                           hipJitOptionInfoLogBuffer, hipJitOptionInfoLogBufferSizeBytes,
                           hipJitOptionLogVerbose };
    void* optionValues[] = { (void*)error_log, (void*)(long)logSize,
                         (void*)info_log, (void*)(long)logSize,
                         (void*)1 };

    hipError_t res = hipModuleLoadDataEx(&cuModule, ptx_code,
                                  sizeof(options)/sizeof(hipJitOption),
                                  options, optionValues);

    if (res != hipSuccess) {
        const char* errStr;
        hipDrvGetErrorString(res, &errStr);
        std::cerr << "Error loading PTX code: " << errStr << "\n";
        std::cerr << "PTX JIT Compilation error log:\n" << error_log << "\n";
        std::cerr << "PTX JIT Compilation info log:\n" << info_log << "\n";
        return -1;
    }


    // Get the kernel function
    res = hipModuleGetFunction(&cuFunction, cuModule, "vector_add_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting kernel function." << std::endl;
        return -1;
    }

    // Vector size
    int N = 1024;
    const int size = N * sizeof(int);

    // Allocate host memory
    std::vector<int> h_A(N, 1); // Vector A initialized to 1
    std::vector<int> h_B(N, 2); // Vector B initialized to 2
    std::vector<int> h_C(N, 0); // Result vector C

    // Allocate device memory
    hipDeviceptr_t d_A, d_B, d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpyHtoD(d_A, h_A.data(), size);
    hipMemcpyHtoD(d_B, h_B.data(), size);
    checkCudaError(hipMemcpyHtoD(d_A, h_A.data(), size), "Error copying h_A to device");
    checkCudaError(hipMemcpyHtoD(d_B, h_B.data(), size), "Error copying h_B to device");

    // Prepare kernel parameters: must match param0..param3 in PTX
    void* args[] = {
        &d_A,       // param0
        &d_B,       // param1
        &d_C,       // param2
        &N          // param3
    };


    // Print input vector A
    std::cout << "Input vector A:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_A[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Print input vector B
    std::cout << "Input vector B:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_B[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Launch the kernel
    int threadsPerBlock = 1;
    int blocksPerGrid = 1;

    hipError_t launchResult = hipModuleLaunchKernel(
        cuFunction,
        blocksPerGrid, 1, 1,       // Grid dimensions (X, Y, Z)
        threadsPerBlock, 1, 1,     // Block dimensions (X, Y, Z)
        0, nullptr,                // Shared memory and stream
        args, nullptr              // Kernel arguments
    );

    std::cout << "Kernel parameters:" << std::endl;
    std::cout << "d_A: " << d_A << std::endl;
    std::cout << "d_B: " << d_B << std::endl;
    std::cout << "d_C: " << d_C << std::endl;
    std::cout << "N: " << N << std::endl;

    if (launchResult != hipSuccess) {
        std::cerr << "Kernel launch failed with error code: " << launchResult << std::endl;
    } else {
        std::cout << "Kernel launched successfully." << std::endl;
    }

    // Copy the result back to the host
    hipMemcpyDtoH(h_C.data(), d_C, size);

    // Print the result
    std::cout << "Result vector C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % 16 == 0) { // Print 16 values per line for readability
            std::cout << std::endl;
        }
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}