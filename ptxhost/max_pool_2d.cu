#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <iomanip>  

// Dummy PTX string placeholder
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry max_pooling_2d_kernel(
	.param .u64 param9,
	.param .u64 param10,
	.param .u64 param11,
	.param .u64 param12,
	.param .u64 param13,
	.param .u64 param14
) {
	.reg .u32 %r1, %r3, %r16, %r17, %r18, %r19, %r20, %r21, %r22, %r25, %r30, %r31, %r32, %r33, %r34, %r35, %r36, %r40, %r42, %r43, %r49, %r54, %r57, %r63, %r67, %r70, %r71, %r72, %r75, %r76, %r80;
	.reg .u64 %rd4, %rd5, %rd6, %rd8, %rd9, %rd10, %rd11, %rd12, %rd13, %rd14, %rd15, %rd23, %rd24, %rd26, %rd27, %rd28, %rd38, %rd39, %rd41, %rd44, %rd45, %rd46, %rd47, %rd48, %rd50, %rd51, %rd55, %rd64, %rd65, %rd66, %rd68, %rd69, %rd73, %rd74, %rd77, %rd78, %rd79, %rd81, %rd82;
	.reg .f32 %f52, %f53, %f59, %f60, %f62;
	.reg .pred %p0, %p2, %p7, %p29, %p37, %p56, %p58, %p61;
max_pooling_2d_kernel_start:
	ld.param.u64 %rd9, [param9];
	ld.param.u64 %rd10, [param10];
	ld.param.u64 %rd11, [param11];
	ld.param.u64 %rd12, [param12];
	ld.param.u64 %rd13, [param13];
	ld.param.u64 %rd14, [param14];
block_16_start:
	setp.lt.u64 %p0, %rd12, %rd13;
	@%p0 bra block_16_end;
	cvt.u32.u64 %r1, %rd14;
	setp.eq.u32 %p2, %r1, 0;
	@%p2 bra block_16_end;
	mov.s32 %r3, -1;
	cvt.s64.s32 %rd4, %r3;
	add.s64 %rd5, %rd13, %rd4;
	cvt.u64.s64 %rd6, %rd5;
	setp.ge.u64 %p7, %rd6, %rd11;
	@%p7 bra block_16_end;
	sub.u64 %rd8, %rd12, %rd13;
	div.u64 %rd15, %rd8, %rd14;
	cvt.u32.u64 %r17, %rd15;
	mov.u32 %r16, %r17;
	mov.u32 %r18, %ntid.y;
	mov.u32 %r19, %ctaid.y;
	mul.lo.s32 %r20, %r18, %r19;
	mov.u32 %r21, %tid.y;
	add.u32 %r22, %r20, %r21;
	mov.u32 %r21, %r22;
	sub.u64 %rd23, %rd11, %rd13;
	div.u64 %rd24, %rd23, %rd14;
	mov.u32 %r25, 1;
	cvt.u64.u32 %rd26, %r25;
	add.u64 %rd27, %rd24, %rd26;
	cvt.u64.u32 %rd28, %r22;
	setp.ge.u64 %p29, %rd28, %rd27;
	@%p29 bra block_16_end;
	mov.u32 %r30, %ntid.x;
	mov.u32 %r31, %ctaid.x;
	mul.lo.s32 %r32, %r30, %r31;
	mov.u32 %r33, %tid.x;
	add.u32 %r34, %r32, %r33;
	mov.u32 %r30, %r34;
	mov.u32 %r35, 1;
	add.u32 %r36, %r16, %r35;
	mov.u32 %r31, %r36;
	setp.ge.u32 %p37, %r34, %r36;
	@%p37 bra block_16_end;
	cvt.u64.u32 %rd38, %r21;
	mul.lo.s64 %rd39, %rd12, %rd38;
	mov.u32 %r40, 2;
	shl.b64 %rd41, %rd39, %r40;
	mov.u32 %r42, 2;
	shl.b32 %r43, %r30, %r42;
	cvt.u64.u32 %rd44, %r43;
	add.u64 %rd45, %rd41, %rd44;
	mul.lo.s64 %rd46, %rd14, %rd45;
	add.u64 %rd47, %rd9, %rd46;
	mov.u64 %rd48, %rd47;
	mov.u32 %r49, 2;
	shl.b64 %rd50, %rd12, %r49;
	mov.u64 %rd51, %rd50;
	mov.f32 %f52, 0fFF800000;
	mov.f32 %f53, %f52;
	mov.u32 %r54, 0;
	cvt.u64.u32 %rd55, %r54;
	mov.u64 %rd11, %rd55;
loop_0_68_start:
block_17_start:
block_18_start:
	setp.eq.u64 %p56, %rd11, %rd13;
	@%p56 bra block_18_end;
	mov.u64 %rd12, %rd13;
	mov.u64 %rd14, %rd48;
loop_0_8_start:
	cvt.u32.u64 %r57, %rd12;
	setp.eq.u32 %p58, %r57, 0;
	@%p58 bra block_17_end;
	ld.global.f32 %f59, [%rd14];
	mov.f32 %f60, %f59;
	setp.gt.f32 %p61, %f60, %f53;
	selp.f32 %f62, %f59, %f53, %p61;
	mov.f32 %f53, %f62;
	mov.s32 %r63, -1;
	cvt.s64.s32 %rd64, %r63;
	add.s64 %rd65, %rd12, %rd64;
	cvt.u64.s64 %rd66, %rd65;
	mov.u64 %rd12, %rd66;
	mov.u32 %r67, 4;
	cvt.u64.u32 %rd68, %r67;
	add.u64 %rd69, %rd14, %rd68;
	mov.u64 %rd14, %rd69;
	bra loop_0_8_start;
loop_0_8_end:
block_18_end:
	mul.lo.s32 %r70, %r31, %r21;
	mov.u32 %r71, 2;
	shl.b32 %r72, %r70, %r71;
	cvt.u64.u32 %rd73, %r72;
	add.u64 %rd74, %rd10, %rd73;
	mov.u32 %r75, 2;
	shl.b32 %r76, %r30, %r75;
	cvt.u64.u32 %rd77, %r76;
	add.u64 %rd78, %rd74, %rd77;
	st.global.f32 [%rd78], %f53;
	bra block_16_end;
block_17_end:
	add.u64 %rd79, %rd48, %rd51;
	mov.u64 %rd48, %rd79;
	mov.u32 %r80, 1;
	cvt.u64.u32 %rd81, %r80;
	add.u64 %rd82, %rd11, %rd81;
	mov.u64 %rd11, %rd82;
	bra loop_0_68_start;
loop_0_68_end:
block_16_end:
max_pooling_2d_kernel_end:
}
)ptx";

// Add this native CUDA kernel implementation
__global__ void native_max_pooling_2d_kernel(float* input, float* output, 
                                            int input_height, int input_width,
                                            int pool_size, int stride) {
    int ox = blockIdx.x * blockDim.x + threadIdx.x;
    int oy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int output_width = (input_width - pool_size) / stride + 1;
    int output_height = (input_height - pool_size) / stride + 1;
    
    if (ox < output_width && oy < output_height) {
        float max_val = -INFINITY;
        
        for (int dy = 0; dy < pool_size; ++dy) {
            for (int dx = 0; dx < pool_size; ++dx) {
                int iy = oy * stride + dy;
                int ix = ox * stride + dx;
                
                float val = input[iy * input_width + ix];
                max_val = fmaxf(max_val, val);
            }
        }
        
        output[oy * output_width + ox] = max_val;
    }
}


const char* getCudaErrorString(hipError_t error) {
    const char* str;
    hipDrvGetErrorString(error, &str);
    return str;
}

float benchmark_pooling(hipCtx_t context, int input_height, int input_width, int pool_size, int stride) {
    hipCtxSetCurrent(context);
    
    // Calculate output dimensions
    int output_height = (input_height - pool_size) / stride + 1;
    int output_width = (input_width - pool_size) / stride + 1;
    
    // Create host data
    std::vector<float> h_input(input_height * input_width);
    std::vector<float> h_output_gpu(output_height * output_width);
    std::vector<float> h_output_cpu(output_height * output_width);
    
    // Initialize input data with random values
    srand(42);  // Fixed seed for reproducibility
    for (auto& val : h_input) {
        val = static_cast<float>(rand()) / RAND_MAX;
    }
    
    // Prepare PTX module
    hipModule_t module;
    hipError_t res = hipModuleLoadData(&module, ptx_code);
    if (res != hipSuccess) {
        std::cerr << "Failed to load PTX module: " << getCudaErrorString(res) << std::endl;
        return 0.0f;
    }
    
    // Get kernel function
    hipFunction_t kernel;
    res = hipModuleGetFunction(&kernel, module, "max_pooling_2d_kernel");
    if (res != hipSuccess) {
        std::cerr << "Failed to get kernel function: " << getCudaErrorString(res) << std::endl;
        hipModuleUnload(module);
        return 0.0f;
    }
    
    // Allocate device memory
    hipDeviceptr_t d_input, d_output;
    res = hipMalloc(&d_input, h_input.size() * sizeof(float));
    if (res != hipSuccess) {
        std::cerr << "Failed to allocate device input memory: " << getCudaErrorString(res) << std::endl;
        hipModuleUnload(module);
        return 0.0f;
    }
    
    res = hipMalloc(&d_output, h_output_gpu.size() * sizeof(float));
    if (res != hipSuccess) {
        std::cerr << "Failed to allocate device output memory: " << getCudaErrorString(res) << std::endl;
        hipFree(d_input);
        hipModuleUnload(module);
        return 0.0f;
    }
    
    // Copy input data to device
    res = hipMemcpyHtoD(d_input, h_input.data(), h_input.size() * sizeof(float));
    if (res != hipSuccess) {
        std::cerr << "Failed to copy input data to device: " << getCudaErrorString(res) << std::endl;
        hipFree(d_output);
        hipFree(d_input);
        hipModuleUnload(module);
        return 0.0f;
    }
    
    // Set up kernel parameters
    // Convert 32-bit integers to 64-bit
    uint64_t input_height_64 = static_cast<uint64_t>(input_height);
    uint64_t input_width_64 = static_cast<uint64_t>(input_width);
    uint64_t pool_size_64 = static_cast<uint64_t>(pool_size);
    uint64_t stride_64 = static_cast<uint64_t>(stride);

    // Use the 64-bit versions in the argument list
    void* args[] = {
        &d_input, &d_output,
        &input_height_64, &input_width_64,
        &pool_size_64, &stride_64
    };
    
    // Set up grid and block dimensions
    int block_size = 16;
    int grid_x = (output_width + block_size - 1) / block_size;
    int grid_y = (output_height + block_size - 1) / block_size;
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreateWithFlags(&start, 0);
    hipEventCreateWithFlags(&stop, 0);
    
    // Record start time
    hipEventRecord(start, 0);
    
    // Launch kernel
    res = hipModuleLaunchKernel(kernel,
                         grid_x, grid_y, 1,  // grid dimensions
                         block_size, block_size, 1,  // block dimensions
                         0, NULL,  // shared memory and stream
                         args, NULL);  // arguments and extra
                         
    if (res != hipSuccess) {
        std::cerr << "Failed to launch kernel: " << getCudaErrorString(res) << std::endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_output);
        hipFree(d_input);
        hipModuleUnload(module);
        return 0.0f;
    }
    
    // Record stop time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    // Calculate elapsed time
    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    
    // Copy result back to host
    res = hipMemcpyDtoH(h_output_gpu.data(), d_output, h_output_gpu.size() * sizeof(float));
    if (res != hipSuccess) {
        std::cerr << "Failed to copy output data from device: " << getCudaErrorString(res) << std::endl;
    }
    
    // Compute reference output on CPU
    for (int oy = 0; oy < output_height; ++oy) {
        for (int ox = 0; ox < output_width; ++ox) {
            float max_val = -INFINITY;
            
            for (int dy = 0; dy < pool_size; ++dy) {
                for (int dx = 0; dx < pool_size; ++dx) {
                    int iy = oy * stride + dy;
                    int ix = ox * stride + dx;
                    
                    float val = h_input[iy * input_width + ix];
                    max_val = fmaxf(max_val, val);
                }
            }
            
            h_output_cpu[oy * output_width + ox] = max_val;
        }
    }
    
    // Validate results
    bool correct = true;
    int mismatch_index = -1;
    float cpu_val = 0.0f, gpu_val = 0.0f;
    
    for (size_t i = 0; i < h_output_gpu.size(); ++i) {
        if (fabsf(h_output_gpu[i] - h_output_cpu[i]) > 1e-5) {
            correct = false;
            mismatch_index = i;
            cpu_val = h_output_cpu[i];
            gpu_val = h_output_gpu[i];
            break;
        }
    }
    
    if (!correct) {
        std::cout << "Mismatch at " << mismatch_index << ": CPU = " 
                  << cpu_val << ", GPU = " << gpu_val << std::endl;
    }
    
    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_output);
    hipFree(d_input);
    hipModuleUnload(module);
    
    /*
    // Output results
    std::cout << "Input: " << input_height << "x" << input_width 
              << " | Pool: " << pool_size << " | Stride: " << stride
              << " | Time: " << elapsed_ms << " ms"
              << " | Correct: " << (correct ? "Yes" : "No") << std::endl;
    */
    return elapsed_ms;
}

float benchmark_cuda_pooling(hipCtx_t context, int input_height, int input_width, int pool_size, int stride) {
    hipCtxSetCurrent(context);
    
    // Calculate output dimensions
    int output_height = (input_height - pool_size) / stride + 1;
    int output_width = (input_width - pool_size) / stride + 1;
    
    // Create host data
    std::vector<float> h_input(input_height * input_width);
    std::vector<float> h_output(output_height * output_width);
    
    // Initialize input data with random values - use same seed as PTX benchmark
    srand(42);  // Fixed seed for reproducibility
    for (auto& val : h_input) {
        val = static_cast<float>(rand()) / RAND_MAX;
    }
    
    // Allocate device memory
    float *d_input, *d_output;
    hipError_t err = hipMalloc(&d_input, h_input.size() * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device input memory: " << hipGetErrorString(err) << std::endl;
        return 0.0f;
    }
    
    err = hipMalloc(&d_output, h_output.size() * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device output memory: " << hipGetErrorString(err) << std::endl;
        hipFree(d_input);
        return 0.0f;
    }
    
    // Copy input data to device
    err = hipMemcpy(d_input, h_input.data(), h_input.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy input data to device: " << hipGetErrorString(err) << std::endl;
        hipFree(d_output);
        hipFree(d_input);
        return 0.0f;
    }
    
    // Set up grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((output_width + blockDim.x - 1) / blockDim.x, 
                 (output_height + blockDim.y - 1) / blockDim.y);
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // Record start time
    hipEventRecord(start);
    
    // Launch kernel
    native_max_pooling_2d_kernel<<<gridDim, blockDim>>>(
        d_input, d_output, input_height, input_width, pool_size, stride
    );
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
        hipEventDestroy(start);
        hipEventDestroy(stop);
        hipFree(d_output);
        hipFree(d_input);
        return 0.0f;
    }
    
    // Record stop time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    // Calculate elapsed time
    float elapsed_ms = 0.0f;
    hipEventElapsedTime(&elapsed_ms, start, stop);
    
    // Copy result back to host
    err = hipMemcpy(h_output.data(), d_output, h_output.size() * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy output data from device: " << hipGetErrorString(err) << std::endl;
    }
    
    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_output);
    hipFree(d_input);
    
    /*
    // Output results
    std::cout << "Native CUDA: " << input_height << "x" << input_width 
              << " | Pool: " << pool_size << " | Stride: " << stride
              << " | Time: " << elapsed_ms << " ms" << std::endl;
    
    */
    return elapsed_ms;
}


int main() {
    // Initialize CUDA
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    
    hipError_t res = hipInit(0);
    if (res != hipSuccess) {
        std::cerr << "Failed to initialize CUDA: " << getCudaErrorString(res) << std::endl;
        return 1;
    }
    
    res = hipDeviceGet(&cuDevice, 0);
    if (res != hipSuccess) {
        std::cerr << "Failed to get CUDA device: " << getCudaErrorString(res) << std::endl;
        return 1;
    }
    
    res = hipCtxCreate(&cuContext, 0, cuDevice);
    if (res != hipSuccess) {
        std::cerr << "Failed to create CUDA context: " << getCudaErrorString(res) << std::endl;
        return 1;
    }

    // Do a warm-up run to initialize CUDA
    std::cout << "Performing warm-up runs..." << std::endl;
    benchmark_pooling(cuContext, 128, 128, 2, 2);
    benchmark_cuda_pooling(cuContext, 128, 128, 2, 2);
    std::cout << "Warm-up complete." << std::endl << std::endl;
    
    // Define test configurations
    struct TestConfig {
        int height, width, pool, stride;
        std::string name;
    };
    
    // Define a range of test configurations with larger dimensions
    TestConfig configs[] = {
        {4096, 4096, 2, 2, "4K image, basic pooling"},
        {8192, 8192, 2, 2, "8K image, basic pooling"},
        {16384, 16384, 2, 2, "16K image, basic pooling"},  // Very large!
        {6144, 6144, 3, 2, "6K image, larger pool"},
        {4096, 8192, 4, 4, "4K×8K image, large pool/stride"},
        {7680, 4320, 3, 1, "8K UHD (7680×4320), stride 1"},
        {2048, 2048, 16, 16, "2K image, extreme pool=stride"}
    };
    
    // Print header with nicer formatting
    std::cout << "╔═══════════════════════════════════════════════════════════════════╗" << std::endl;
    std::cout << "║           PERFORMANCE COMPARISON: PTX vs Native CUDA              ║" << std::endl;
    std::cout << "╠═════════════════════╦═══════════════╦═══════════════╦════════════╣" << std::endl;
    std::cout << "║ Configuration        ║ PTX Time (ms) ║ CUDA Time (ms)║  Speedup   ║" << std::endl;
    std::cout << "╠═════════════════════╬═══════════════╬═══════════════╬════════════╣" << std::endl;

    // Run benchmarks 
    for (const auto& config : configs) {
        // Run each benchmark 3 times and take the best time
        float ptx_time = 1000.0f; // Start with a high value
        float cuda_time = 1000.0f;
        
        for (int i = 0; i < 3; i++) {
            // Run PTX benchmark
            float run_time = benchmark_pooling(cuContext, 
                                          config.height, config.width, 
                                          config.pool, config.stride);
            ptx_time = std::min(ptx_time, run_time);
            
            // Run native CUDA benchmark
            run_time = benchmark_cuda_pooling(cuContext,
                                           config.height, config.width, 
                                           config.pool, config.stride);
            cuda_time = std::min(cuda_time, run_time);
        }
        
        // Calculate speedup ratio
        float speedup = 0.0f;
        if (cuda_time > 0.0001f) {
            speedup = ptx_time / cuda_time;
        }
        
        // Prepare the configuration string
        std::string config_str = std::to_string(config.height) + "×" + 
                                 std::to_string(config.width) + " P" + 
                                 std::to_string(config.pool) + " S" + 
                                 std::to_string(config.stride);
        
        // Print results with aligned columns
        std::cout << "║ " << std::left << std::setw(19) << config_str 
                  << " ║ " << std::right << std::setw(13) << std::fixed << std::setprecision(3) << ptx_time
                  << " ║ " << std::right << std::setw(13) << std::fixed << std::setprecision(3) << cuda_time
                  << " ║ " << std::right << std::setw(10) << std::fixed << std::setprecision(2);
                  
        if (speedup > 0.0f) {
            std::cout << speedup << "× ";
        } else {
            std::cout << "N/A  ";
        }
        std::cout << "║" << std::endl;
    }
    
    std::cout << "╚═════════════════════╩═══════════════╩═══════════════╩════════════╝" << std::endl;
    
    // Clean up
    hipCtxDestroy(cuContext);
    
    return 0;
}