#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// PTX code as a string
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry vector_add_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r7, %r8, %r17, %r21, %r6, %r26, %r36, %r32, %r9, %r24, %r23, %r4, %r35, %r20, %r27, %r12, %r11, %r22, %r5;
    .reg .u64 %rd2, %rd0, %rd16, %rd19, %rd29, %rd28, %rd34, %rd31, %rd33, %rd14, %rd18, %rd13, %rd30, %rd3, %rd15, %rd1;
    .reg .pred %p10, %p25;
    vector_add_kernel_start:
    ld.param.u64 %rd0, [param9];
    ld.param.u64 %rd1, [param10];
    ld.param.u64 %rd2, [param11];
    ld.param.u64 %rd3, [param12];
    block_1_start:
    mov.u32 %r4, %ntid.x;
    mov.u32 %r5, %ctaid.x;
    mul.lo.s32 %r6, %r4, %r5;
    mov.u32 %r7, %tid.x;
    add.s32 %r8, %r6, %r7;
    mov.u32 %r5, %r8;
    cvt.u32.u64 %r9, %rd3;
    setp.ge.u32 %p10, %r8, %r9;
    @%p10 bra block_1_end;
    mov.u32 %r11, 2;
    shl.b32 %r12, %r5, %r11;
    mov.u32 %r5, %r12;
    cvt.u64.u32 %rd13, %r12;
    add.s64 %rd14, %rd2, %rd13;
    cvt.u64.u32 %rd15, %r5;
    add.s64 %rd16, %rd1, %rd15;
    ld.global.u32 %r17, [%rd16];
    cvt.u64.u32 %rd18, %r5;
    add.s64 %rd19, %rd0, %rd18;
    ld.global.u32 %r20, [%rd19];
    add.s32 %r21, %r17, %r20;
    st.global.u32 [%rd14], %r21;
    block_1_end:
    mul.lo.s32 %r22, %r4, %r5;
    add.s32 %r23, %r22, %r7;
    mov.u32 %r5, %r23;
    cvt.u32.u64 %r24, %rd3;
    setp.ge.u32 %p25, %r23, %r24;
    @%p25 bra vector_add_kernel_end;
    mov.u32 %r26, 2;
    shl.b32 %r27, %r5, %r26;
    mov.u32 %r5, %r27;
    cvt.u64.u32 %rd28, %r27;
    add.s64 %rd29, %rd2, %rd28;
    cvt.u64.u32 %rd30, %r5;
    add.s64 %rd31, %rd1, %rd30;
    ld.global.u32 %r32, [%rd31];
    cvt.u64.u32 %rd33, %r5;
    add.s64 %rd34, %rd0, %rd33;
    ld.global.u32 %r35, [%rd34];
    add.s32 %r36, %r32, %r35;
    st.global.u32 [%rd29], %r36;
    vector_add_kernel_end:
}

.visible .entry vector_add_loop_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12
) {
    .reg .u32 %r7, %r43, %r46, %r6, %r40, %r34, %r44, %r47, %r28, %r31, %r45, %r24, %r9, %r4, %r32, %r15, %r27, %r13, %r8, %r18, %r17, %r14, %r29, %r16, %r5, %r12, %r10, %r30;
    .reg .u64 %rd39, %rd42, %rd0, %rd26, %rd25, %rd20, %rd2, %rd3, %rd22, %rd1, %rd21, %rd41, %rd36, %rd37, %rd23, %rd38;
    .reg .pred %p19, %p35, %p33;
    vector_add_loop_kernel_start:
    ld.param.u64 %rd0, [param9];
    ld.param.u64 %rd1, [param10];
    ld.param.u64 %rd2, [param11];
    ld.param.u64 %rd3, [param12];
    mov.u32 %r4, %ntid.x;
    mov.u32 %r5, %ntid.z;
    mul.lo.s32 %r6, %r4, %r5;
    mov.u32 %r7, %ntid.y;
    mul.lo.s32 %r8, %r6, %r7;
    mov.u32 %r7, %r8;
    mov.u32 %r9, 2;
    shl.b32 %r10, %r8, %r9;
    mov.u32 %r11, %r10;
    mov.u32 %r12, %ctaid.x;
    mul.lo.s32 %r13, %r4, %r12;
    mov.u32 %r14, %tid.x;
    add.s32 %r15, %r13, %r14;
    mov.u32 %r5, %r15;
    mov.u32 %r16, 2;
    shl.b32 %r17, %r15, %r16;
    mov.u32 %r4, %r17;
    loop_0_start:
    cvt.u32.u64 %r18, %rd3;
    setp.ltu.u32 %p19, %r5, %r18;
    @%p19 bra loop_0_start;
    mov.u64 return_value_3, %rd3;
    mov.u64 return_value_2, %rd2;
    mov.u64 return_value_1, %rd1;
    mov.u64 return_value_0, %rd0;
    ret;
    cvt.u64.u32 %rd20, %r4;
    add.s64 %rd21, %rd2, %rd20;
    cvt.u64.u32 %rd22, %r4;
    add.s64 %rd23, %rd1, %rd22;
    ld.global.u32 %r24, [%rd23];
    cvt.u64.u32 %rd25, %r4;
    add.s64 %rd26, %rd0, %rd25;
    ld.global.u32 %r27, [%rd26];
    add.s32 %r28, %r24, %r27;
    st.global.u32 [%rd21], %r28;
    add.s32 %r30, %r4, %r29;
    mov.u32 %r4, %r30;
    add.s32 %r31, %r5, %r7;
    mov.u32 %r5, %r31;
    bra loop_0_start;
    loop_0_end:
    block_2_start:
    cvt.u32.u64 %r32, %rd3;
    setp.ltu.u32 %p33, %r5, %r32;
    @%p33 bra block_2_end;
    ret;
    block_2_end:
    cvt.u32.u64 %r34, %rd3;
    setp.ltu.u32 %p35, %r5, %r34;
    @%p35 bra vector_add_loop_kernel_end;
    ret;
    cvt.u64.u32 %rd36, %r4;
    add.s64 %rd37, %rd2, %rd36;
    cvt.u64.u32 %rd38, %r4;
    add.s64 %rd39, %rd1, %rd38;
    ld.global.u32 %r40, [%rd39];
    cvt.u64.u32 %rd41, %r4;
    add.s64 %rd42, %rd0, %rd41;
    ld.global.u32 %r43, [%rd42];
    add.s32 %r44, %r40, %r43;
    st.global.u32 [%rd37], %r44;
    add.s32 %r46, %r4, %r45;
    mov.u32 %r4, %r46;
    add.s32 %r47, %r5, %r7;
    mov.u32 %r5, %r47;
    bra vector_add_loop_kernel_end;
    vector_add_loop_kernel_end:
}


)ptx";

hipError_t checkCudaError(hipError_t result, const char* message) {
    if (result != hipSuccess) {
        const char* errorString;
        hipDrvGetErrorString(result, &errorString);
        std::cerr << message << ": " << errorString << std::endl;
    }
    return result;
}


int main() {
    // Initialize CUDA
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;

    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);

    // Load the PTX code as a module
    hipError_t res = hipModuleLoadData(&cuModule, ptx_code);
    if (res != hipSuccess) {
        std::cerr << "Error loading PTX code." << std::endl;
        return -1;
    }

    // Get the kernel function
    res = hipModuleGetFunction(&cuFunction, cuModule, "vector_add_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting kernel function." << std::endl;
        return -1;
    }

    // Vector size
    int N = 1024;
    const int size = N * sizeof(int);

    // Allocate host memory
    std::vector<int> h_A(N, 1); // Vector A initialized to 1
    std::vector<int> h_B(N, 2); // Vector B initialized to 2
    std::vector<int> h_C(N, 0); // Result vector C

    // Allocate device memory
    hipDeviceptr_t d_A, d_B, d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data to device
    hipMemcpyHtoD(d_A, h_A.data(), size);
    hipMemcpyHtoD(d_B, h_B.data(), size);
    checkCudaError(hipMemcpyHtoD(d_A, h_A.data(), size), "Error copying h_A to device");
    checkCudaError(hipMemcpyHtoD(d_B, h_B.data(), size), "Error copying h_B to device");

    // Prepare kernel parameters: must match param0..param3 in PTX
    void* args[] = {
        &d_A,       // param0
        &d_B,       // param1
        &d_C,       // param2
        &N          // param3
    };


    // Print input vector A
    std::cout << "Input vector A:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_A[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Print input vector B
    std::cout << "Input vector B:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_B[i] << " ";
        if ((i + 1) % 16 == 0) {
            std::cout << std::endl;
        }
    }

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    hipError_t launchResult = hipModuleLaunchKernel(
        cuFunction,
        blocksPerGrid, 1, 1,       // Grid dimensions
        threadsPerBlock, 1, 1,     // Block dimensions
        0, nullptr,                // Shared memory and stream
        args, nullptr              // Kernel arguments
    );


    std::cout << "Kernel parameters:" << std::endl;
    std::cout << "d_A: " << d_A << std::endl;
    std::cout << "d_B: " << d_B << std::endl;
    std::cout << "d_C: " << d_C << std::endl;
    std::cout << "N: " << N << std::endl;

    if (launchResult != hipSuccess) {
        std::cerr << "Kernel launch failed with error code: " << launchResult << std::endl;
    } else {
        std::cout << "Kernel launched successfully." << std::endl;
    }

    // Copy the result back to the host
    hipMemcpyDtoH(h_C.data(), d_C, size);

    // Print the result
    std::cout << "Result vector C:" << std::endl;
    for (int i = 0; i < N; ++i) {
        std::cout << h_C[i] << " ";
        if ((i + 1) % 16 == 0) { // Print 16 values per line for readability
            std::cout << std::endl;
        }
    }

    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}