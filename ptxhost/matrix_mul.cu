#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// PTX code as a string (replace with your actual matrix_mul_kernel PTX)
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry matrix_mul_kernel(
	.param .u64 param9,
	.param .u64 param10,
	.param .u64 param11,
	.param .u64 param12,
	.param .u64 param13,
	.param .u64 param14
) {
	.reg .u32 %r0, %r1, %r2, %r3, %r4, %r5, %r8, %r15, %r16, %r17, %r18, %r21, %r23, %r24, %r30, %r34, %r38, %r41, %r42, %r45, %r49, %r50, %r51, %r52, %r53;
	.reg .u64 %rd6, %rd9, %rd10, %rd11, %rd12, %rd13, %rd14, %rd19, %rd22, %rd25, %rd26, %rd27, %rd28, %rd29, %rd31, %rd32, %rd33, %rd36, %rd37, %rd39, %rd40, %rd43, %rd44, %rd46, %rd47, %rd48, %rd54, %rd55, %rd56;
	.reg .pred %p7, %p20, %p35;
matrix_mul_kernel_start:
	ld.param.u64 %rd9, [param9];
	ld.param.u64 %rd10, [param10];
	ld.param.u64 %rd11, [param11];
	ld.param.u64 %rd12, [param12];
	ld.param.u64 %rd13, [param13];
	ld.param.u64 %rd14, [param14];
block_11_start:
	mov.u32 %r0, %ntid.y;
	mov.u32 %r1, %ctaid.y;
	mul.lo.s32 %r2, %r0, %r1;
	mov.u32 %r3, %tid.y;
	add.u32 %r4, %r2, %r3;
	mov.u32 %r5, %r4;
	cvt.u64.u32 %rd6, %r4;
	setp.ge.u64 %p7, %rd6, %rd12;
	@%p7 bra block_11_end;
	mov.u32 %r8, %ntid.x;
	mov.u32 %r15, %ctaid.x;
	mul.lo.s32 %r16, %r8, %r15;
	mov.u32 %r17, %tid.x;
	add.u32 %r18, %r16, %r17;
	mov.u32 %r15, %r18;
	cvt.u64.u32 %rd19, %r18;
	setp.ge.u64 %p20, %rd19, %rd14;
	@%p20 bra block_11_end;
	mov.u32 %r21, 2;
	shl.b64 %rd22, %rd14, %r21;
	mov.u64 %rd12, %rd22;
	mov.u32 %r23, 2;
	shl.b32 %r24, %r15, %r23;
	cvt.u64.u32 %rd25, %r24;
	add.u64 %rd26, %rd10, %rd25;
	mov.u64 %rd27, %rd26;
	cvt.u64.u32 %rd28, %r5;
	mul.lo.s64 %rd29, %rd28, %rd13;
	mov.u32 %r30, 2;
	shl.b64 %rd31, %rd29, %r30;
	add.u64 %rd32, %rd9, %rd31;
	mov.u64 %rd33, %rd32;
	mov.u32 %r34, 0;
	mov.u32 %r3, %r34;
loop_0_38_start:
block_12_start:
	setp.ne.u64 %p35, %rd13, 0;
	@%p35 bra block_12_end;
	cvt.u64.u32 %rd36, %r5;
	mul.lo.s64 %rd37, %rd36, %rd14;
	mov.u32 %r38, 2;
	shl.b64 %rd39, %rd37, %r38;
	add.u64 %rd40, %rd11, %rd39;
	mov.u32 %r41, 2;
	shl.b32 %r42, %r15, %r41;
	cvt.u64.u32 %rd43, %r42;
	add.u64 %rd44, %rd40, %rd43;
	st.global.u32 [%rd44], %r3;
	bra block_11_end;
block_12_end:
	mov.s32 %r45, -1;
	cvt.s64.s32 %rd46, %r45;
	add.s64 %rd47, %rd13, %rd46;
	cvt.u64.s64 %rd48, %rd47;
	mov.u64 %rd13, %rd48;
	ld.global.u32 %r49, [%rd27];
	ld.global.u32 %r50, [%rd33];
	mul.lo.s32 %r51, %r49, %r50;
	add.u32 %r52, %r51, %r3;
	mov.u32 %r3, %r52;
	mov.u32 %r53, 4;
	cvt.u64.u32 %rd54, %r53;
	add.u64 %rd55, %rd33, %rd54;
	mov.u64 %rd33, %rd55;
	add.u64 %rd56, %rd27, %rd12;
	mov.u64 %rd27, %rd56;
	bra loop_0_38_start;
loop_0_38_end:
block_11_end:
matrix_mul_kernel_end:
}

)ptx";

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>

__global__ void cuda_matrix_mul_global(const int* A, const int* B, int* C, int M, int N, int P) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < P) {
        int sum = 0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * P + col];
        }
        C[row * P + col] = sum;
    }
}

void benchmark(int N) {
    std::vector<int> h_A(N * N), h_B(N * N), h_C_ptx(N * N), h_C_cuda(N * N);
    for (int& val : h_A) val = rand() % 10;
    for (int& val : h_B) val = rand() % 10;

    int *d_A_cuda, *d_B_cuda, *d_C_cuda;
    hipMalloc(&d_A_cuda, N * N * sizeof(int));
    hipMalloc(&d_B_cuda, N * N * sizeof(int));
    hipMalloc(&d_C_cuda, N * N * sizeof(int));
    hipMemcpy(d_A_cuda, h_A.data(), N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B_cuda, h_B.data(), N * N * sizeof(int), hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + threads.x - 1) / threads.x, (N + threads.y - 1) / threads.y);

    // Warm-up CUDA kernel
    cuda_matrix_mul_global<<<blocks, threads>>>(d_A_cuda, d_B_cuda, d_C_cuda, N, N, N);
    hipDeviceSynchronize();

    // Timed CUDA kernel run
    hipEvent_t start_cuda, stop_cuda;
    hipEventCreate(&start_cuda);
    hipEventCreate(&stop_cuda);
    hipEventRecord(start_cuda);
    cuda_matrix_mul_global<<<blocks, threads>>>(d_A_cuda, d_B_cuda, d_C_cuda, N, N, N);
    hipEventRecord(stop_cuda);
    hipEventSynchronize(stop_cuda);

    float time_cuda = 0;
    hipEventElapsedTime(&time_cuda, start_cuda, stop_cuda);
    hipMemcpy(h_C_cuda.data(), d_C_cuda, N * N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_A_cuda);
    hipFree(d_B_cuda);
    hipFree(d_C_cuda);
    hipEventDestroy(start_cuda);
    hipEventDestroy(stop_cuda);

    // PTX setup
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    hipInit(0);
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);

    char error_log[8192];
    char info_log[8192];
    hipJitOption options[] = {
        hipJitOptionErrorLogBuffer, hipJitOptionErrorLogBufferSizeBytes,
        hipJitOptionInfoLogBuffer, hipJitOptionInfoLogBufferSizeBytes,
        hipJitOptionLogVerbose
    };
    void* optionValues[] = {
        (void*)error_log, (void*)(uintptr_t)sizeof(error_log),
        (void*)info_log,  (void*)(uintptr_t)sizeof(info_log),
        (void*)1
    };

    hipError_t res = hipModuleLoadDataEx(&cuModule, ptx_code, 5, options, optionValues);
    if (res != hipSuccess) {
        std::cerr << "Failed to load PTX module.\nError log:\n" << error_log << "\n";
        return;
    }

    res = hipModuleGetFunction(&cuFunction, cuModule, "matrix_mul_kernel");
    if (res != hipSuccess) {
        std::cerr << "Failed to get PTX function.\n";
        return;
    }

    hipDeviceptr_t d_A, d_B, d_C;
    hipMalloc(&d_A, N * N * sizeof(int));
    hipMalloc(&d_B, N * N * sizeof(int));
    hipMalloc(&d_C, N * N * sizeof(int));
    hipMemcpyHtoD(d_A, h_A.data(), N * N * sizeof(int));
    hipMemcpyHtoD(d_B, h_B.data(), N * N * sizeof(int));

    uint64_t m64 = N, n64 = N, p64 = N;
    void* args[] = { &d_A, &d_B, &d_C, &m64, &n64, &p64 };

    // Warm-up PTX kernel
    hipModuleLaunchKernel(cuFunction,
                   (N + 15) / 16, (N + 15) / 16, 1,
                   16, 16, 1,
                   0, 0, args, 0);
    hipCtxSynchronize();

    // Timed PTX kernel run
    hipEvent_t start_ptx, stop_ptx;
    hipEventCreateWithFlags(&start_ptx, hipEventDefault);
    hipEventCreateWithFlags(&stop_ptx, hipEventDefault);
    hipEventRecord(start_ptx, 0);
    hipModuleLaunchKernel(cuFunction,
                   (N + 15) / 16, (N + 15) / 16, 1,
                   16, 16, 1,
                   0, 0, args, 0);
    hipEventRecord(stop_ptx, 0);
    hipEventSynchronize(stop_ptx);

    float time_ptx = 0;
    hipEventElapsedTime(&time_ptx, start_ptx, stop_ptx);

    hipMemcpyDtoH(h_C_ptx.data(), d_C, N * N * sizeof(int));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    
    // After getting both CUDA and PTX results, add validation:
    
    // Check if results are all zeros (common error case)
    bool ptx_all_zeros = true;
    bool cuda_all_zeros = true;
    
    // Check first few elements (for efficiency)
    int check_limit = std::min(100, N * N);
    for (int i = 0; i < check_limit; i++) {
        if (h_C_ptx[i] != 0) ptx_all_zeros = false;
        if (h_C_cuda[i] != 0) cuda_all_zeros = false;
        if (!ptx_all_zeros && !cuda_all_zeros) break;
    }
    
    // Validate results - use CUDA as reference implementation
    bool ptx_matches_cuda = true;
    int mismatch_i = -1, mismatch_j = -1;
    int ptx_val = 0, cuda_val = 0;
    
    // Sample the matrix (stride increases with size for better performance)
    int stride = N > 1000 ? N / 100 : 1;
    for (int i = 0; i < N && ptx_matches_cuda; i += stride) {
        for (int j = 0; j < N && ptx_matches_cuda; j += stride) {
            int idx = i * N + j;
            if (h_C_ptx[idx] != h_C_cuda[idx]) {
                ptx_matches_cuda = false;
                mismatch_i = i;
                mismatch_j = j;
                ptx_val = h_C_ptx[idx];
                cuda_val = h_C_cuda[idx];
            }
        }
    }
    
    // Output results with validation information
    std::cout << "N = " << N 
              << " | PTX: " << time_ptx << " ms"
              << " | CUDA: " << time_cuda << " ms"
              << " | Validation: ";
    
    if (cuda_all_zeros) {
        std::cout << "CUDA produced all zeros (likely incorrect)";
    } else if (ptx_all_zeros) {
        std::cout << "PTX produced all zeros (likely incorrect)";
    } else if (ptx_matches_cuda) {
        std::cout << "PASS (PTX matches CUDA)";
    } else {
        std::cout << "FAIL - mismatch at [" << mismatch_i << "][" << mismatch_j 
                  << "]: CUDA=" << cuda_val << ", PTX=" << ptx_val;
                  
        // For small matrices, print full comparison when there's a mismatch
        if (N <= 8) {
            std::cout << "\nFull comparison for small matrix:" << std::endl;
            for (int i = 0; i < N; i++) {
                for (int j = 0; j < N; j++) {
                    int idx = i * N + j;
                    std::cout << "(" << h_C_cuda[idx] << "," << h_C_ptx[idx] << ") ";
                }
                std::cout << std::endl;
            }
        }
    }
    std::cout << std::endl;
}


int main() {
    srand(time(0));
    for (int N = 4; N <= 40000; N *= 2)
        benchmark(N);
    return 0;
}
