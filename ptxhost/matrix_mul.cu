#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// PTX code as a string (replace with your actual matrix_mul_kernel PTX)
const char* ptx_code = R"ptx(
.version 8.0
.target sm_80
.visible .entry matrix_mul_kernel(
    .param .u64 param9,
    .param .u64 param10,
    .param .u64 param11,
    .param .u64 param12,
    .param .u64 param13,
    .param .u64 param14
) {
    .reg .u32 %r1, %r45, %r38, %r34, %r49, %r52, %r24, %r3, %r51, %r16, %r30, %r53, %r42, %r23, %r8, %r2, %r50, %r4, %r18, %r41, %r21, %r17, %r15, %r0, %r5;
    .reg .u64 %rd46, %rd55, %rd6, %rd29, %rd9, %rd22, %rd36, %rd12, %rd39, %rd32, %rd13, %rd19, %rd25, %rd10, %rd40, %rd54, %rd33, %rd28, %rd37, %rd48, %rd44, %rd56, %rd27, %rd43, %rd47, %rd14, %rd31, %rd11, %rd26;
    .reg .pred %p20, %p7, %p35;
    matrix_mul_kernel_start:
    ld.param.u64 %rd9, [param9];
    ld.param.u64 %rd10, [param10];
    ld.param.u64 %rd11, [param11];
    ld.param.u64 %rd12, [param12];
    ld.param.u64 %rd13, [param13];
    ld.param.u64 %rd14, [param14];
    block_3_start:
    mov.u32 %r0, %ntid.y;
    mov.u32 %r1, %ctaid.y;
    mul.lo.s32 %r2, %r0, %r1;
    mov.u32 %r3, %tid.y;
    add.u32 %r4, %r2, %r3;
    mov.u32 %r5, %r4;
    cvt.u64.u32 %rd6, %r4;
    setp.ge.u64 %p7, %rd6, %rd12;
    @%p7 bra block_3_end;
    mov.u32 %r8, %ntid.x;
    mov.u32 %r15, %ctaid.x;
    mul.lo.s32 %r16, %r8, %r15;
    mov.u32 %r17, %tid.x;
    add.u32 %r18, %r16, %r17;
    mov.u32 %r15, %r18;
    cvt.u64.u32 %rd19, %r18;
    setp.ge.u64 %p20, %rd19, %rd14;
    @%p20 bra block_3_end;
    mov.u32 %r21, 2;
    shl.b64 %rd22, %rd14, %r21;
    mov.u64 %rd12, %rd22;
    mov.u32 %r23, 2;
    shl.b32 %r24, %r15, %r23;
    cvt.u64.u32 %rd25, %r24;
    add.u64 %rd26, %rd10, %rd25;
    mov.u64 %rd27, %rd26;
    cvt.u64.u32 %rd28, %r5;
    mul.lo.s64 %rd29, %rd28, %rd13;
    mov.u32 %r30, 2;
    shl.b64 %rd31, %rd29, %r30;
    add.u64 %rd32, %rd9, %rd31;
    mov.u64 %rd33, %rd32;
    mov.u32 %r34, 0;
    mov.u32 %r3, %r34;
    loop_0_start:
    block_4_start:
    setp.ne.u64 %p35, %rd13, 0;
    @%p35 bra block_4_end;
    cvt.u64.u32 %rd36, %r5;
    mul.lo.s64 %rd37, %rd36, %rd14;
    mov.u32 %r38, 2;
    shl.b64 %rd39, %rd37, %r38;
    add.u64 %rd40, %rd11, %rd39;
    mov.u32 %r41, 2;
    shl.b32 %r42, %r15, %r41;
    cvt.u64.u32 %rd43, %r42;
    add.u64 %rd44, %rd40, %rd43;
    st.global.u32 [%rd44], %r3;
    bra block_3_end;
    block_4_end:
    mov.s32 %r45, -1;
    cvt.s64.s32 %rd46, %r45;
    add.s64 %rd47, %rd13, %rd46;
    cvt.u64.s64 %rd48, %rd47;
    mov.u64 %rd13, %rd48;
    ld.global.u32 %r49, [%rd27];
    ld.global.u32 %r50, [%rd33];
    mul.lo.s32 %r51, %r49, %r50;
    add.u32 %r52, %r51, %r3;
    mov.u32 %r3, %r52;
    mov.u32 %r53, 4;
    cvt.u64.u32 %rd54, %r53;
    add.u64 %rd55, %rd33, %rd54;
    mov.u64 %rd33, %rd55;
    add.u64 %rd56, %rd27, %rd12;
    mov.u64 %rd27, %rd56;
    bra loop_0_start;
    loop_0_end:
    block_3_end:
    matrix_mul_kernel_end:
}


)ptx";

hipError_t checkCudaError(hipError_t result, const char* message) {
    if (result != hipSuccess) {
        const char* errorString;
        hipDrvGetErrorString(result, &errorString);
        std::cerr << message << ": " << errorString << std::endl;
    }
    return result;
}

int main() {
    // Matrix dimensions
    int m = 4, n = 4, p = 4;
    const int sizeA = m * n;
    const int sizeB = n * p;
    const int sizeC = m * p;

	
	int h_A[4][4] = {
    {1, 2, 3, 4},
    {5, 6, 7, 8},
    {9, 10, 11, 12},
    {13, 14, 15, 16}
	};

	int h_B[4][4] = {
		{16, 15, 14, 13},
		{12, 11, 10, 9},
		{8, 7, 6, 5},
		{4, 3, 2, 1}
	};

	int h_C[4][4] = {0}; // Initialize result matrix with zeros


    // Device pointers
    hipDeviceptr_t d_A, d_B, d_C;
    hipInit(0);
    hipDevice_t cuDevice;
    hipCtx_t cuContext;
    hipModule_t cuModule;
    hipFunction_t cuFunction;
    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&cuContext, 0, cuDevice);

    const unsigned int logSize = 8192;
    char error_log[logSize];
    char info_log[logSize];

    hipJitOption options[] = { hipJitOptionErrorLogBuffer, hipJitOptionErrorLogBufferSizeBytes,
                           hipJitOptionInfoLogBuffer, hipJitOptionInfoLogBufferSizeBytes,
                           hipJitOptionLogVerbose };
    void* optionValues[] = { (void*)error_log, (void*)(long)logSize,
                         (void*)info_log, (void*)(long)logSize,
                         (void*)1 };

    hipError_t res = hipModuleLoadDataEx(&cuModule, ptx_code,
                                  sizeof(options)/sizeof(hipJitOption),
                                  options, optionValues);

    if (res != hipSuccess) {
        const char* errStr;
        hipDrvGetErrorString(res, &errStr);
        std::cerr << "Error loading PTX code: " << errStr << "\n";
        std::cerr << "PTX JIT Compilation error log:\n" << error_log << "\n";
        std::cerr << "PTX JIT Compilation info log:\n" << info_log << "\n";
        return -1;
    }

	
	// Flatten h_A and h_B for device transfer
	int flat_A[16];
	int flat_B[16];
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j < 4; ++j) {
			flat_A[i * 4 + j] = h_A[i][j];
			flat_B[i * 4 + j] = h_B[i][j];
		}
	}

		
	
	// Allocate device memory
	hipMalloc(&d_A, 16 * sizeof(int));
	hipMalloc(&d_B, 16 * sizeof(int));
	hipMalloc(&d_C, 16 * sizeof(int));

	// Copy data to device
	hipMemcpyHtoD(d_A, flat_A, 16 * sizeof(int));
	hipMemcpyHtoD(d_B, flat_B, 16 * sizeof(int));

    // Load PTX and get kernel
    res = hipModuleLoadData(&cuModule, ptx_code);
    if (res != hipSuccess) {
        std::cerr << "Error loading PTX code." << std::endl;
        return -1;
    }
    res = hipModuleGetFunction(&cuFunction, cuModule, "matrix_mul_kernel");
    if (res != hipSuccess) {
        std::cerr << "Error getting kernel function." << std::endl;
        return -1;
    }

    // Prepare kernel parameters
    uint64_t m64 = m, n64 = n, p64 = p;
    void* args[] = { &d_A, &d_B, &d_C, &m64, &n64, &p64 };

    // Launch configuration
    int threadsPerBlockX = 16, threadsPerBlockY = 16;
    int blocksPerGridX = (p + threadsPerBlockX - 1) / threadsPerBlockX;
    int blocksPerGridY = (m + threadsPerBlockY - 1) / threadsPerBlockY;

    hipError_t launchResult = hipModuleLaunchKernel(
        cuFunction,
        blocksPerGridX, blocksPerGridY, 1,
        threadsPerBlockX, threadsPerBlockY, 1,
        0, nullptr,
        args, nullptr
    );

    if (launchResult != hipSuccess) {
        std::cerr << "Kernel launch failed with error code: " << launchResult << std::endl;
    } else {
        std::cout << "Kernel launched successfully." << std::endl;
    }

    // Copy result back
    //hipMemcpyDtoH(h_C.data(), d_C, sizeC * sizeof(int));


	int flat_C[16];
	hipMemcpyDtoH(flat_C, d_C, 16 * sizeof(int));

	
	// Reconstruct h_C from flat_C
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j < 4; ++j) {
			h_C[i][j] = flat_C[i * 4 + j];
		}
	}

	std::cout << "Result matrix C:" << std::endl;
	for (int i = 0; i < 4; ++i) {
		for (int j = 0; j < 4; ++j) {
			std::cout << h_C[i][j] << " ";
		}
		std::cout << std::endl;
	}


    // Clean up
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipModuleUnload(cuModule);
    hipCtxDestroy(cuContext);

    return 0;
}